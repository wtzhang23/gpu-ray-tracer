#include "rayenv/scene.h"
namespace renv {
Scene::Scene(Canvas& canvas, Camera camera, rprimitives::Texture atlas, std::vector<rprimitives::Hitable*> hitables): 
                                            canvas(canvas), cam(camera), atlas(atlas) {
    if (!hitables.empty()) {
        hipMallocManaged(&this->hitables, sizeof(rprimitives::Hitable*) * hitables.size());
        hipMemcpy(this->hitables, hitables.data(), sizeof(rprimitives::Hitable*) * hitables.size(), hipMemcpyHostToDevice);
    }
}
}