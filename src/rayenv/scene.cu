#include "rayenv/scene.h"
namespace renv {
Scene::Scene(Canvas& canvas, Camera camera, rprimitives::Texture atlas, std::vector<rprimitives::Hitable*> hitables, rprimitives::VertexBuffer buffer): 
                                            canvas(canvas), cam(camera), atlas(atlas), buffer(buffer), nh(hitables.size()) {
    if (!hitables.empty()) {
        hipMallocManaged(&this->hitables, sizeof(rprimitives::Hitable*) * hitables.size());
        hipMemcpy(this->hitables, hitables.data(), sizeof(rprimitives::Hitable*) * hitables.size(), hipMemcpyHostToDevice);
    }
}
}