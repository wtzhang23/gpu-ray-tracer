#include "hip/hip_runtime.h"
#include "rayenv/gpu/scene.h"
#include "rayenv/gpu/scene.cuh"
#include "rayenv/transformation.h"
#include "rayprimitives/material.h"
#include "rayprimitives/gpu/material.cuh"
#include "rayprimitives/gpu/hitable.cuh"
#include "rayopt/bounding_box.h"

namespace renv {
namespace gpu {
static const int MAX_DEPTH = 10;

__device__
bool cast_local(Scene* scene, const rmath::Ray<float>& r, rprimitives::Isect& isect, const Transformation& t) {
    rprimitives::gpu::Hitable** hitables = scene->get_hitables();
    rprimitives::gpu::Hitable* h = hitables[t.get_hitable_idx()];
    rmath::Vec3<float> local_dir = t.vec_to_local(r.direction());
    float dir_len = local_dir.len();
    rmath::Ray<float> local_ray = rmath::Ray<float>({t.point_to_local(r.origin()), local_dir});
    bool rv = h->hit(local_ray, scene, isect);
    if (rv) {
        isect.norm = t.vec_from_local(isect.norm);
        isect.time *= dir_len;
    }
    return rv;
}

__device__
bool cast_ray(Scene* scene, const rmath::Ray<float>& r, rprimitives::Isect& isect) {
    Environment& env = scene->get_environment();
    Transformation* trans = env.get_trans();
    bool hit = false;
    const ropt::gpu::BVH& bvh = scene->get_bvh();
    if (bvh.empty()) {
        for (int i = 0; i < env.n_trans(); i++) {
            const Transformation& t = trans[i];
            hit |= cast_local(scene, r, isect, t);
        }
    } else {
        ropt::gpu::BVHIterator iter{r, INFINITY, scene};
        while (iter.current() >= 0) {
            const Transformation& t = trans[iter.current()];
            if (cast_local(scene, r, isect, t)) {
                hit = true;
            }
            iter.next(INFINITY);
        }
        if (env.is_debugging()) {
            printf("tested %d / %d bounding boxes for %d objs\n", 
                    iter.n_intersections(), iter.max_intersections(), env.n_trans());
        }
    }
    return hit;
}

enum FrameType {
    NORMAL,
    REFLECT,
    REFRACT
};

struct RayFrame {
    rmath::Ray<float> ray;
    rmath::Vec3<float> hit_pt;
    rmath::Vec3<float> norm;
    rmath::Vec4<float> atten;
    rprimitives::Material* last_mat;
    float last_eta;
    FrameType type;
    int depth;
    bool in_obj;
};

__device__
rmath::Vec4<float> propagate_ray(Scene* scene, const rmath::Ray<float>& r, rprimitives::Isect& isect) {
    renv::Environment& env = scene->get_environment();

    RayFrame frames[MAX_DEPTH];
    frames[0] = {r, rmath::Vec3<float>(), rmath::Vec3<float>(), rmath::Vec4<float>({1.0f, 1.0f, 1.0f, 1.0f}), 
                        NULL, 1.0f, FrameType::NORMAL, env.get_recurse_depth(), false};
    int stack_top = 0;
    rmath::Vec4<float> acc_col{};
    
    while (stack_top >= 0) {
        RayFrame& top = frames[stack_top];
        switch (top.type) {
            case FrameType::NORMAL: {
                isect.time = INFINITY; // reset
                if (env.is_debugging()) {
                    printf("shooting a ray\n");
                }
                if (cast_ray(scene, top.ray, isect)) {
                    acc_col += top.atten * rprimitives::gpu::illuminate(top.ray, isect, scene);
                    if (top.depth > 0) {
                        if (top.in_obj) {
                            rmath::Vec4<float> kt = top.last_mat->get_Kt();
                            float ar = pow(isect.time, kt[0]);
                            float ag = pow(isect.time, kt[1]);
                            float ab = pow(isect.time, kt[2]);
                            float aa = pow(isect.time, kt[3]);
                            top.atten *= rmath::Vec4<float>({ar, ag, ab, aa});
                        }
                        frames[stack_top].type = FrameType::REFLECT;
                        frames[stack_top].hit_pt = top.ray.at(isect.time);
                        frames[stack_top].last_mat = isect.mat;
                        top.norm = isect.norm;
                    } else {
                        stack_top--;
                    }
                } else {
                    stack_top--;
                }
                break;
            }
            case FrameType::REFLECT: {
                rmath::Vec4<float> kr = isect.mat->get_Kr();
                frames[stack_top].type = FrameType::REFRACT;
                if (kr[0] > 0.0f || kr[1] > 0.0f || kr[2] > 0.0f || kr[3] > 0.0f) {
                    if (env.is_debugging()) {
                        printf("preparing to shoot a reflection ray\n");
                    }
                    stack_top++;
                    RayFrame& new_top = frames[stack_top];
                    new_top.type = FrameType::NORMAL;
                    new_top.last_mat = top.last_mat;
                    new_top.in_obj = top.in_obj;
                    new_top.atten = top.atten * kr;
                    new_top.depth = top.depth - 1;
                    rmath::Vec3<float> reflect_dir = rmath::reflect(top.ray.direction(), top.norm);
                    new_top.ray = rmath::Ray<float>(top.hit_pt, reflect_dir);
                }
                break;
            }
            case FrameType::REFRACT: {
                rmath::Vec4<float> kt = isect.mat->get_Kt();
                if (kt[0] > 0.0f || kt[1] > 0.0f || kt[2] > 0.0f || kt[3] > 0.0f) {
                    if (env.is_debugging()) {
                        printf("preparing to shoot a refraction ray\n");
                    }
                    top.type = FrameType::NORMAL;
                    bool tir;
                    rmath::Vec3<float> refract_dir = rmath::refract(top.ray.direction(), top.norm, 
                                        top.last_eta, top.last_mat->get_eta(), tir);
                    if (tir) {
                        stack_top--;
                    } else {
                        top.ray = rmath::Ray<float>(top.hit_pt, refract_dir);
                        top.in_obj = !top.in_obj;
                        top.depth--;
                        if (top.in_obj) {
                            top.last_eta = top.last_mat->get_eta();
                        } else {
                            top.last_eta = 1.0f;
                        }
                    }
                } else {
                    stack_top--;
                }
                break;
            }
        }
    }
    return acc_col;
}
}
}