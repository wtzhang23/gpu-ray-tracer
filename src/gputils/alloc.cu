#include "gputils/alloc.h"
#include <cassert>

namespace gputils {
void* create_buffer(const int n_data, const int data_size) {
    void* buffer;
    auto rv = hipMallocManaged(&buffer, data_size * n_data);
    assert(rv == 0);
    return buffer;
}

void free_buffer(void* buffer) {
    auto rv = hipFree(buffer);
    assert(rv == 0);
}
}