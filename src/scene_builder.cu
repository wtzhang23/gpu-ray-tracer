#include "hip/hip_runtime.h"
#include "scene_builder.h"
#include <thrust/scan.h>
#include "raymath/linear.h"
#include "iostream"
#include "rayenv/scene.h"
#include "rayenv/scene.cuh"
#include "rayprimitives/texture.cuh"
#include "rayprimitives/material.h"
#include "rayprimitives/material.cuh"
#include "rayprimitives/hitable.cuh"
#include "rayprimitives/trimesh.cuh"
#include "rayprimitives/light.cuh"
#include "gputils/alloc.h"
#include "assets.h"

namespace rtracer {

std::vector<rmath::Vec3<float>> generate_normals(const std::vector<rmath::Vec3<float>>& vertices,
                                                    const std::vector<rmath::Vec3<int>>& triangles) {
    std::vector<rmath::Vec3<float>> normals = std::vector<rmath::Vec3<float>>(vertices.size());
    for (const rmath::Vec3<int>& tri : triangles) {
        rmath::Vec3<float> a = vertices[tri[1]] - vertices[tri[0]];
        rmath::Vec3<float> b = vertices[tri[2]] - vertices[tri[0]];
        rmath::Vec3<float> n = rmath::cross(a, b).normalized();
        normals[tri[0]] += n;
        normals[tri[1]] += n;
        normals[tri[2]] += n;
    }

    // renormalize sums
    for (rmath::Vec3<float>& n : normals) {
        n = n.normalized();
    }
    return normals;
}

struct MeshConfig {
    rprimitives::Trimesh** meshes;
    int* ends;
    rmath::Vec3<int>* indices;
    rprimitives::Material* mats;
    rprimitives::Shade* shadings;
    rmath::Vec3<float>* mesh_pos;
    rmath::Quat<float>* mesh_rot;
    int n_meshes;
};

__global__
void build_meshes(MeshConfig* config) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < config->n_meshes; i += stride) {
        int begin = i == 0 ? 0 : config->ends[i - 1];
        int count = config->ends[i] - begin;
        rprimitives::TriInner* triangles = new rprimitives::TriInner[count];
        for (int j = 0; j < count; j++) {
            rprimitives::TriInner inner = rprimitives::TriInner(config->indices[begin + j], config->mats[begin + j], config->shadings[begin + j]);
            triangles[j] = inner;
        }
        rprimitives::Trimesh* mesh = new rprimitives::Trimesh(triangles, count);
        mesh->set_position(config->mesh_pos[i]);
        mesh->set_orientation(config->mesh_rot[i]);
        config->meshes[i] = mesh;
    } 
}

struct LightConfig {
    rprimitives::Light** lights;
    rmath::Vec3<float>* point_light_pos;
    rmath::Vec3<float>* dir_light_dir;
    rmath::Vec4<float>* point_light_col;
    rmath::Vec4<float>* dir_light_col;
    int n_points;
    int n_directional;
};

__global__
void build_lights(LightConfig* config) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < config->n_points + config->n_directional; i += stride) {
        rprimitives::Light* light;
        if (i > config->n_points) {
            rprimitives::PointLight* point_light = new rprimitives::PointLight();
            point_light->set_color(config->point_light_col[i]);
            point_light->set_pos(config->point_light_pos[i]);
            light = point_light;
        } else {
            int j = i - config->n_points;
            rprimitives::DirLight* dir_light = new rprimitives::DirLight();
            dir_light->set_color(config->dir_light_col[j]);
            dir_light->set_shine_dir(config->dir_light_dir[j]);
            light = dir_light;
        }
        config->lights[i] = light;
    }
}

renv::Scene* SceneBuilder::build_scene(renv::Canvas canvas, renv::Camera camera) {
    // load assets
    gputils::TextureBuffer4D<float> atlas = assets::read_png(atlas_path.c_str());

    // flatten meshes
    std::vector<rmath::Vec3<int>> flattened_triangles{};
    std::vector<rprimitives::Shade> flattened_shadings{};
    std::vector<rprimitives::Material> flattened_mats{};
    std::vector<rmath::Vec3<float>> flattened_mesh_pos{};
    std::vector<rmath::Quat<float>> flattened_mesh_rot{};
    std::vector<int> counts{};
    for (MeshBuilder& b : this->meshes) {
        assert(b.triangles.size() == b.shadings.size());
        assert(b.triangles.size() == b.mats.size());
        counts.push_back(b.triangles.size());
        for (int i = 0; i < b.triangles.size(); i++) {
            flattened_triangles.push_back(b.triangles[i]);
            flattened_shadings.push_back(b.shadings[i]);
            flattened_mats.push_back(b.mats[i]);
        }
        flattened_mesh_pos.push_back(b.pos);
        flattened_mesh_rot.push_back(b.rot);
    }

    // build vertex buffer
    std::vector<rmath::Vec3<float>> normals = generate_normals(vertices, flattened_triangles);
    rprimitives::VertexBuffer buffer{vertices, normals};

    // build meshes
    thrust::inclusive_scan(counts.data(), counts.data() + counts.size(), counts.data());
    int* ends = gputils::copy_to_gpu<int>(counts.data(), counts.size());
    rprimitives::Shade* dev_shadings = gputils::copy_to_gpu<rprimitives::Shade>(flattened_shadings.data(), 
                                                                                flattened_shadings.size());
    rprimitives::Material* dev_mats = gputils::copy_to_gpu<rprimitives::Material>(flattened_mats.data(),
                                                                                flattened_mats.size());
    rmath::Vec3<int>* dev_tris = gputils::copy_to_gpu<rmath::Vec3<int>>(flattened_triangles.data(), 
                                                                                flattened_triangles.size());
    rmath::Vec3<float>* dev_mesh_pos = gputils::copy_to_gpu<rmath::Vec3<float>>(flattened_mesh_pos.data(), 
                                                                                flattened_mesh_pos.size());
    rmath::Quat<float>* dev_mesh_rot = gputils::copy_to_gpu<rmath::Quat<float>>(flattened_mesh_rot.data(), 
                                                                                flattened_mesh_rot.size());
    rprimitives::Trimesh** hitables = (rprimitives::Trimesh**) gputils::create_buffer(counts.size(), sizeof(rprimitives::Trimesh*));
    int n_hitables = this->meshes.size();
    MeshConfig mesh_config = {
                hitables, 
                ends, 
                dev_tris, 
                dev_mats, 
                dev_shadings, 
                dev_mesh_pos,
                dev_mesh_rot,
                (int) counts.size(),
            };
    MeshConfig* mesh_config_ptr = gputils::copy_to_gpu(&mesh_config, 1);
    build_meshes<<<1, 512>>>(mesh_config_ptr);
    hipFree(mesh_config_ptr);
    hipFree(ends);
    hipFree(dev_shadings);
    hipFree(dev_mats);
    hipFree(dev_tris);

    // create lights
    rmath::Vec3<float>* dev_point_light_pos = gputils::copy_to_gpu<rmath::Vec3<float>>(point_light_pos.data(), point_light_pos.size());
    rmath::Vec3<float>* dev_dir_light_dir = gputils::copy_to_gpu<rmath::Vec3<float>>(dir_light_dir.data(), dir_light_dir.size());
    rmath::Vec4<float>* dev_point_light_col = gputils::copy_to_gpu<rmath::Vec4<float>>(point_light_col.data(), point_light_col.size());
    rmath::Vec4<float>* dev_dir_light_col = gputils::copy_to_gpu<rmath::Vec4<float>>(dir_light_col.data(), dir_light_col.size());
    int n_point_lights = point_light_col.size();
    int n_dir_lights = dir_light_col.size();
    int n_lights = n_point_lights + n_dir_lights;
    rprimitives::Light** lights = (rprimitives::Light**) gputils::create_buffer(n_lights, sizeof(rprimitives::Light*));
    LightConfig light_config = {lights, dev_point_light_pos, dev_dir_light_dir, dev_point_light_col, dev_dir_light_col, n_point_lights, n_dir_lights};
    LightConfig* light_config_ptr = gputils::copy_to_gpu(&light_config, 1);
    build_lights<<<1, 1024>>>(light_config_ptr);
    hipFree(dev_point_light_pos);
    hipFree(dev_dir_light_dir);
    hipFree(dev_point_light_col);
    hipFree(dev_dir_light_col);
    hipFree(light_config_ptr);

    // configure local scene
    renv::Scene local_scene = renv::Scene{canvas, camera, atlas, (rprimitives::Hitable**) hitables,
                                        n_hitables, lights, n_lights, buffer};
    local_scene.set_ambience(ambience);
    local_scene.set_recurse_depth(recurse_depth);
    renv::Scene* s = (renv::Scene*) gputils::create_buffer(1, sizeof(renv::Scene));
    hipMemcpy(s, &local_scene, sizeof(renv::Scene), hipMemcpyDefault);
    return s;
}

void SceneBuilder::build_cube(float scale, rmath::Vec3<float> pos, rmath::Quat<float> rot, 
                                            rprimitives::Shade shade, rprimitives::Material mat) {
    /*   e-----f
     *  /|    /|
     * a-----b |
     * | g---|-h
     * |/    |/
     * c-----d
     */
    const rmath::Vec3<float> _a{-0.5f, 0.5f, -0.5f};
    const rmath::Vec3<float> _b{0.5f, 0.5f, -0.5f};
    const rmath::Vec3<float> _c{-0.5f, -0.5f, -0.5f};
    const rmath::Vec3<float> _d{0.5f, -0.5f, -0.5f};
    const rmath::Vec3<float> _e{-0.5f, 0.5f, 0.5f};
    const rmath::Vec3<float> _f{0.5f, 0.5f, 0.5f};
    const rmath::Vec3<float> _g{-0.5f, -0.5f, 0.5f};
    const rmath::Vec3<float> _h{0.5f, -0.5f, 0.5f};
    rmath::Vec3<float> a = scale * _a;
    rmath::Vec3<float> b = scale * _b;
    rmath::Vec3<float> c = scale * _c;
    rmath::Vec3<float> d = scale * _d;
    rmath::Vec3<float> e = scale * _e;
    rmath::Vec3<float> f = scale * _f;
    rmath::Vec3<float> g = scale * _g;
    rmath::Vec3<float> h = scale * _h;

    MeshBuilder& builder = create_mesh(pos, rot);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(a), add_vertex(b), add_vertex(d)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(c), add_vertex(d), add_vertex(a)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(a), add_vertex(b), add_vertex(e)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(e), add_vertex(f), add_vertex(b)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(d), add_vertex(h), add_vertex(b)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(b), add_vertex(f), add_vertex(h)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(c), add_vertex(g), add_vertex(a)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(a), add_vertex(e), add_vertex(g)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(g), add_vertex(h), add_vertex(e)},
                            shade, mat);
    builder.add_triangle(rmath::Vec3<int>{add_vertex(e), add_vertex(f), add_vertex(h)},
                            shade, mat);
}

}