#include "hip/hip_runtime.h"
#include "rayprimitives/texture.h"
#include "rayprimitives/texture.cuh"
#include "raymath/linear.h"

namespace rprimitives {
__device__
rmath::Vec4<float> get_color_from_texture(Texture& text, float text_x, float text_y) {
    float4 col = tex2D<float4>(text.get_buffer().get_obj(), text_x, text_y);
    return rmath::Vec4<float>({col.x, col.y, col.z, col.w});
}

__device__
rmath::Vec4<float> get_color_from_texture(Texture& text, float u, float v, int text_x, int text_y, int text_width, int text_height) {
    return get_color_from_texture(text, text_x + u * text_width, text_y + v * text_height);
}

rmath::Vec4<float> Texture::color_to_vec(renv::Color c) {
    return 1.0f / UINT8_MAX * rmath::Vec4<float>({(float) c.r(), (float) c.g(), (float) c.b(), (float) c.a()});
}

void Texture::free(Texture& texture) {
    gputils::TextureBuffer4D<float>::free(texture.get_buffer());
}
}