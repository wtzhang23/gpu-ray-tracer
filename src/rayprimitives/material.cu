#include "hip/hip_runtime.h"
#include "rayprimitives/material.cuh"
#include "rayenv/scene.h"
#include "rayprimitives/light.cuh"

namespace rprimitives {

__device__
rmath::Vec4<float> illuminate(const rmath::Ray<float>& org_ray, const Isect& isect, renv::Scene* s) {
    rprimitives::Light** lights = s->get_lights();
    rmath::Vec4<float> summed_colors = isect.mat->get_Ke() + isect.mat->get_Ka() * s->get_ambience();
    for (int i = 0; i < s->n_lights(); i++) {
        rprimitives::Light* light = lights[i];
        rmath::Vec3<float> dir_to_light;
        rmath::Vec4<float> incoming_light = light->shine(org_ray.at(isect.time), dir_to_light, s);
        
        // diffuse component
        float norm_dot = abs(rmath::dot(dir_to_light, isect.norm));
        rmath::Vec4<float> diffuse = norm_dot * isect.mat->get_Kd();
        
        // specular component
        rmath::Vec3<float> reflected = -rmath::reflect(-dir_to_light, isect.norm);
        float reflect_dot = rmath::dot(reflected, org_ray.direction());
        rmath::Vec4<float> specular = pow(max(reflect_dot, 0.0f), isect.mat->get_alpha()) * isect.mat->get_Ks();
        summed_colors += (diffuse + specular) * incoming_light;
    }
    return summed_colors;
}

}