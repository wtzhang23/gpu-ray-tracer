#include "hip/hip_runtime.h"
#include "linear.h"
#include "raytracer.h"
#include "iostream"

namespace raytracer {
    __global__
    void color_green(scene::Scene scene) {
        canvas::Canvas& canvas = scene.get_canvas();
        int height = canvas.get_height();
        int width = canvas.get_width();
        
        int idx = blockDim.x * blockIdx.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int i = idx; i < height * width; i += stride) {
            int row = i / width;
            int col = i % width;
            canvas.set_color(row, col, canvas::Color(0.5, 1.0, 0.5));
        }
    }

    void update_scene(scene::Scene& scene) {
        color_green<<<1024, 1024>>>(scene);
        int rv = hipDeviceSynchronize();
        assert(rv == 0);
        canvas::Canvas& canvas = scene.get_canvas();
    }
}