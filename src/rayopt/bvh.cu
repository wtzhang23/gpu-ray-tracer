#include "hip/hip_runtime.h"
#include "rayopt/gpu/bvh.h"
#include "rayopt/bounding_box.h"
#include "rayopt/z_order.h"
#include "raymath/linear.h"
#include "rayenv/gpu/scene.h"
#include <thrust/sort.h>

namespace ropt {
namespace gpu {

__global__
void gen_numbers(int* arr, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        arr[i] = i;
    }
}

__global__
void gen_morton(unsigned long* codes, BoundingBox* boxes, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n; i += stride) {
        if (boxes[i].is_degenerate()) {
            codes[i] = ULONG_MAX;
        } else {
            rmath::Vec3<float> center = -boxes[i].center(); // negate since first bit indicates pos/neg
            codes[i] = ropt::z_order(center);
        }
    }
}

__global__
void reorder(int* ordering, BoundingBox* from_boxes, BoundingBox* to_boxes, int n_objs) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < n_objs; i += stride) {
        to_boxes[i] = from_boxes[ordering[i]];
    }    
}

__global__
void build_bvh_layer(BoundingBox* boxes, int batch_size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    BoundingBox* from = boxes;
    while (batch_size >= 2) {
        int offset = idx * batch_size;
        int next_offset = offset >> 1;
        int next_addr = blockDim.x * gridDim.x * batch_size;
        assert(offset < next_addr);
        BoundingBox* to = &from[next_addr];
        for (int i = 0; i < (batch_size >> 1); i++) {
            BoundingBox left = from[offset + 2 * i];
            BoundingBox right = from[offset + 2 * i + 1];
            to[next_offset + i] = ropt::merge(left, right);
        }
        from = to;
        batch_size >>= 1;
    }
}

const int BVH_THREADS = 512;
__host__
void build_bvh(BoundingBox* flattened_tree, int n_boxes) {
    if (n_boxes >= BVH_THREADS * 2) {
        int n_blocks = n_boxes / (2 * BVH_THREADS);
        build_bvh_layer<<<n_blocks, BVH_THREADS>>>(flattened_tree, 2); // reduce one layer
        build_bvh(flattened_tree + (n_blocks * BVH_THREADS * 2), n_boxes / 2);
    } else {
        build_bvh_layer<<<1, 1>>>(flattened_tree, n_boxes);
    }
}
BVH::BVH(BoundingBox* org_boxes, int n_org_objs): n_objs(n_org_objs) {
    int n_blocks = (n_org_objs + BVH_THREADS - 1) / BVH_THREADS;
    unsigned long* codes;
    BoundingBox* flattened_tree;
    int rv = hipMalloc(&ordering, n_org_objs * sizeof(int));
    assert(rv == 0);
    rv = hipMalloc(&codes, n_org_objs * sizeof(unsigned long));
    assert(rv == 0);
    rv = hipMalloc(&flattened_tree, 2 * n_org_objs * sizeof(BoundingBox));
    assert(rv == 0);
    gen_morton<<<n_blocks, BVH_THREADS>>>(codes, org_boxes, n_org_objs);
    gen_numbers<<<n_blocks, BVH_THREADS>>>(ordering, n_org_objs);
    thrust::sort_by_key(thrust::device, codes, codes + n_org_objs, ordering);
    reorder<<<n_blocks, BVH_THREADS>>>(ordering, org_boxes, flattened_tree, n_objs);
    build_bvh(flattened_tree, n_org_objs);
    this->boxes = flattened_tree;
    hipFree(codes);
}

void BVH::free(BVH& b) {
    hipFree(b.boxes);
    hipFree(b.ordering);
}

CUDA_HOSTDEV
void BVHIterator::step_up() {
    while (node_idx % 2 == 1) {
        node_idx = parent();
    }

    if (node_idx == 0) {
        return;
    } else {
        node_idx = parent();
        assert(node_idx >= 1);
        node_idx = right_child();
        assert(node_idx >= 1);
    }
}

CUDA_HOSTDEV
void BVHIterator::step_next() {
    int lc = left_child();
    if (lc < 0) {
        step_up();
    } else {
        node_idx = lc;
    }
}

CUDA_HOSTDEV
BVHIterator::BVHIterator(const rmath::Ray<float>& r, renv::gpu::Scene* scene): 
            bvh(scene->get_bvh()), r(r), node_idx(1), scene(scene) {}

CUDA_HOSTDEV
int BVHIterator::current() const {
    if (node_idx == 0) {
        return -1;
    }
    int box_idx = get_box_idx(node_idx);
    if (box_idx > bvh.n_objs) {
        return -1;
    }
    int after_order = bvh.ordering[box_idx];
    return after_order;
}
CUDA_HOSTDEV
BoundingBox BVHIterator::cur_bounding_box() const {
    if (node_idx == 0) {
        return BoundingBox{};
    }
    int box_idx = get_box_idx(node_idx);
    return bvh.boxes[box_idx];
}

CUDA_HOSTDEV
bool BVHIterator::intersects_node() const {
    if (node_idx == 0) {
        return false;
    }
    float time;
    return cur_bounding_box().intersects(r, time);
}

}
}