#include "canvas.h"
#include <cassert>

namespace canvas {
int* create_buffer(int width, int height) {
    int* buffer;
    auto rv = hipMallocManaged(&buffer, sizeof(int) * width * height);
    assert(rv == 0);
    return buffer;
}

void free_buffer(int* buffer) {
    auto rv = hipFree(buffer);
    assert(rv == 0);
}
}